
#include <hip/hip_runtime.h>
#include<stdio.h>

void helloCPU()
{
    printf("good cpu.");
}

__global__ void helloGPU()
{
    printf("good gpu.");
}

int main()
{
    helloCPU();
    helloGPU<<<2, 1>>>();
    hipDeviceSynchronize();
}