
#include <hip/hip_runtime.h>
#include<stdio.h>

void cpuFunction()
{
    printf("This is called on CPU.\n");
}

__global__
void firstParallel()
{
    if (threadIdx.x == 1023 && blockIdx.x == 255)
        printf("sucess.\n");
}

__global__
void multidimGrid()
{
    printf("block:(%i,%i,%i), thread:(%i,%i,%i)\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
    cpuFunction();
    firstParallel<<<256, 1024>>>();
    dim3 num_of_block(4, 4, 1);
    dim3 num_of_thread(2, 4, 2);
    multidimGrid<<<num_of_block, num_of_thread>>>();
    hipDeviceSynchronize();
}