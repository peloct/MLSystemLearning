
#include <hip/hip_runtime.h>
#include<stdio.h>

void init(int* a, const int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

__global__
void doubleElements(int* a, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        a[idx] *= 2;
}

bool checkElements(int* a, const int N)
{
    for (int i = 0; i < N; ++i)
        if (a[i] != i * 2)
            return false;
    return true;
}

int main()
{
    int N = 1000;
    int* a;

    size_t size = N * sizeof(int);

    hipMallocManaged(&a, size);

    init(a, N);

    size_t threads_per_block = 256;
    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();

    printf("Is doubled? : %s\n", checkElements(a, N) ? "True" : "False");
}