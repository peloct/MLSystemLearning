#include "hip/hip_runtime.h"
#include<pybind11/pybind11.h>

namespace py = pybind11;

void init(int* a, const int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

__global__
void doubleElements(int* a, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        a[idx] *= 2;
}

bool checkElements(int* a, const int N)
{
    for (int i = 0; i < N; ++i)
        if (a[i] != i * 2)
            return false;
    return true;
}

bool run(int n)
{
    int N = n;
    int* a;

    size_t size = N * sizeof(int);

    hipMallocManaged(&a, size);

    init(a, N);

    size_t threads_per_block = 256;
    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();

    return checkElements(a, N);
}

// PYBIND11_MODULE 에서 앞에 인자의 이름은
// 모듈 이름과 같아야 한다.

PYBIND11_MODULE(cmake_test, m){
    m.doc() = "pybind11 example plugin";
    m.def("run", &run, "A cuda function that double all elem",
        py::arg("n")=1000);
    m.attr("int_var") = 40;
    m.attr("str_var") = py::cast("hello");
}